#include "hip/hip_runtime.h"
#include "fft.h"
#include "hip/hip_runtime.h"

__device__ unsigned char getr(float x) {
    return (tanh((x - 0.375f) * 6.0f) + 1.0f) * 127.0f;
}
__device__ unsigned char getg(float x) {
    return (tanh((x - 0.6250f) * 6.0f) + 1.0f) * 127.0f;
}
__device__ unsigned char getb(float x) {
    return (exp(-20.0f * (x - 0.25f) * (x - 0.25f) - 2.0f * exp(-(x + 0.05f) * (x + 0.05f) * 144.0f)) * 0.5f + 1.0f + tanh((x - 0.875f) * 6.0f)) * 127.0f;
}
__global__ void imgfill(float2* d_k, uchar3* d_img,int size)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int imgx, imgy;
    imgx = (x >= size / 2) ? x - size / 2 : x + size / 2;
    imgy = (y >= size / 2) ? y - size / 2 : y + size / 2;
    float2 k = d_k[y * size + x];
    float in = k.x * k.x + k.y * k.y;
    in = log(in * (1.0f / 256.0f/size) + 0.8f) * 0.07f;
    uchar3 c;
    c.x = getb(in);
    c.y = getg(in);
    c.z = getr(in);
    d_img[imgy * size + imgx] = c;
}

__global__ void fill(float2* d_x, uchar3* d_8uc3,int size,int w,int h) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int imgx, imgy;
    float cx, cy;
    unsigned char r;
    if (x >= size / 2 + w / 2) {
        imgx = 0;
        cx = size - x;
        cx = exp(-cx * cx * (1.0f / 1024.0f));
    }
    else if (x < size / 2 + w / 2 && x >= w) {
        imgx = w - 1;
        cx = x - w;
        cx = exp(-cx * cx * (1.0f / 1024.0f));
    }
    else {
        imgx = x;
        cx = 1.0f;
    }

    if (y >= size / 2 + h / 2) {
        imgy = 0;
        cy = size - y;
        cy = exp(-cy * cy * (1.0f / 1024.0f));
    }
    else if (y < size / 2 + h / 2 && y >= h) {
        imgy = h - 1;
        cy = y - h;
        cy = exp(-cy * cy * (1.0f / 1024.0f));
    }
    else {
        imgy = y;
        cy = 1.0f;
    }
    r = d_8uc3[imgy * w + imgx].x;
    d_x[y * size + x].x = r * cx * cy;
    d_x[y * size + x].y = 0;
}


void fft_tranformer(uchar3 * d_8uc3,float2 * d_x,float2 * d_k,
                    uchar3 *d_img,hipfftHandle *fftPlan,
                    unsigned char * pframe,unsigned  char * pDst,
                    int width,int height,int size)
{
    hipMemcpy(d_8uc3, pframe, width * height * 3, hipMemcpyHostToDevice);
    fill << < dim3(size / 128, size, 1), dim3(128, 1, 1) >> > (d_x, d_8uc3,size,width,height);
    hipfftExecC2C(*fftPlan, d_x, d_k, HIPFFT_FORWARD);
    imgfill << < dim3(size / 128, size, 1), dim3(128, 1, 1) >> > (d_k, d_img,size);
    hipMemcpy(pDst, d_img, size * size * 3, hipMemcpyDeviceToHost);
}